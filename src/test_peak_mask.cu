#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
//  gpRepel : An R package for GPU computing - testing the moving average function
//  COMPILE: nvcc -arch sm_11 test_*.cu
//
//  This program is free software; you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation; version 3 of the License.
//
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program; if not, write to the Free Software
//  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
//
//  Author: Piotr Wąsiewicz pwasiewi@gmail.com
//
////////////////////////////////////////////////////////////////////////////////////////////////////

#include "gpRepel.h"


using namespace std;

typedef thrust::tuple<Numeric,Numeric> 								Numeric2;
typedef typename thrust::device_vector<Numeric>::iterator         	NumericIterator;
typedef typename thrust::tuple<NumericIterator, NumericIterator>  	NumericIteratorTuple;
typedef typename thrust::zip_iterator<NumericIteratorTuple>       	Numeric2Iterator;
typedef thrust::tuple<Numeric,Numeric,Numeric> 						Numeric3;
typedef typename thrust::tuple<NumericIterator, NumericIterator, NumericIterator>  NumericIteratorTuple3;
typedef typename thrust::zip_iterator<NumericIteratorTuple3>       	Numeric3Iterator;

////////////////////////////////////////////////////////////////////////////////////////////////////
// VecReorder - the gpu functor implementing the dot product between 3d vectors
////////////////////////////////////////////////////////////////////////////////////////////////////
struct VecReorder : public thrust::binary_function<Numeric2,Numeric2,Numeric>
{
    Numeric w, maxb;
    Numeric ai,bi,av,bv, result;

    __host__ __device__
    VecReorder(Numeric w, Numeric maxb) : w(w), maxb(maxb) {}
    __host__ __device__
        Numeric operator()(const Numeric2& a, const Numeric2& b) const
        {
            Numeric ai=(int) thrust::get<0>(a) % (int) maxb;
            Numeric av=thrust::get<1>(a);
            Numeric bi=(int) thrust::get<0>(b) % (int) maxb;
            Numeric bv=thrust::get<1>(b);
	    	int lastone = (int) maxb*((int) thrust::get<0>(a) / (int) maxb)-1;
	    	if(ai > bi)
            	return thrust::get<0>(a);
	    	else
	    		return lastone;
        }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// minus_and_divide_zip - gpu functor implementing the dot product between 3d vectors
////////////////////////////////////////////////////////////////////////////////////////////////////
struct  minus_and_divide_zip : public thrust::binary_function<Numeric3,Numeric3,Numeric>
{
    Numeric w, maxb;
    Numeric ai,bi,av,bv, result;

    __host__ __device__
    minus_and_divide_zip(Numeric w, Numeric maxb) : w(w), maxb(maxb) {}
    __host__ __device__
    Numeric operator()(const Numeric3& a, const Numeric3& b) const
    {
		Numeric ai=(int) thrust::get<0>(a) % (int) maxb;
		Numeric av=thrust::get<1>(a);
		Numeric bi=(int) thrust::get<0>(b) % (int) maxb;
		Numeric bv=thrust::get<1>(b);
	    int lastone = (int) maxb*((int) thrust::get<0>(a) / (int) maxb);
	    if(ai > bi)
                return (av - bv)/w;
	    else
	    	if((int)w - 1 != (int) ai)
	    		return (thrust::get<2>(b) - thrust::get<1>(b)) / ((int)w - (int)ai -1);
	    	else
	    		return 0;
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// simple_moving_average - GPU function of the simple average with a window w points forward, 
// after a given point; idata, vout - input and output matrices with m (rows) x n (cols) dimensions,
// 
////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename InputVector, typename OutputVector>
void simple_moving_average(size_t m, size_t n, const InputVector& idata, size_t w, OutputVector& vout)
{
    typedef typename InputVector::value_type T;

    if (idata.size() < w)
        return;
    thrust::device_vector<size_t> output(m*n);
    thrust::device_vector<Numeric> voutput(m*n);
    thrust::device_vector<Numeric> data(m*n);
    thrust::device_vector<Numeric> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end());

    thrust::inclusive_scan(idata.begin(), idata.end(), data.begin());

    Numeric2Iterator first = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin(), data.begin()));
    Numeric2Iterator firstw = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin() + w, data.begin() + w));
    Numeric2Iterator last  = thrust::make_zip_iterator(thrust::make_tuple(vindex.end(),   data.end()));

    thrust::transform(firstw, last, first, output.begin(), VecReorder(w,m));
 
    thrust::gather(output.begin(), output.end(), data.begin(), voutput.begin());

    Numeric3Iterator first3 = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin(), data.begin(), voutput.begin()));
    Numeric3Iterator firstw3 = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin() + w, data.begin() + w, voutput.begin() + w));
    Numeric3Iterator last3  = thrust::make_zip_iterator(thrust::make_tuple(vindex.end(), data.end(), voutput.end()));

    thrust::transform(firstw3, last3, first3, vout.begin(), minus_and_divide_zip(w,m));
    thrust::fill(vout.end()-w,vout.end(),vout[vout.size()-w-1]);
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// print an array m x n with vectors in columns
////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T>
void print(size_t m, size_t n, thrust::device_vector<T>& d_data)
{
    thrust::host_vector<T> h_data = d_data;

    for(size_t i = 0; i < n; i++)
    {
        for(size_t j = 0; j < m; j++)
            std::cout << " " << h_data[j + i * m] << " ";
        std::cout << "\n";
    }
    std::cout << "\n";
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// print an array m x n with vectors in columns as one vector: one after another
////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T>
void printvec(size_t m, size_t n, thrust::device_vector<T>& d_data)
{
    thrust::host_vector<T> h_data = d_data;

    for(size_t i = 0; i < m*n; i++)
        std::cout << setw(4) << h_data[i] << " ";
    std::cout << "\n";
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// mask01 - the gpu functor of making mask, where values greater than w are set to 1
//
template <typename T>
struct mask01 : public thrust::unary_function<T,T>
{
    T w;
	__host__ __device__
    mask01(T w) : w(w) {}

	__host__ __device__
    T operator()(const T& a) const
    {
    	if(a > w)
    		return 1;
    	else
    		return 0;
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// segmented by peaks sums 
// 
template <typename InputVector, typename OutputVector>
void segmented_peak_sums(size_t m, size_t n, const InputVector& idata, OutputVector& data)
{
  typedef typename InputVector::value_type T;

  thrust::device_vector<Numeric> peakmask(m*n);

  thrust::transform(idata.begin(), idata.end(), peakmask.begin(), mask01<Numeric>(Integer(0)));
  //cout << "peakmask after fill ():" << endl;
  //printvec(1,NDATA,peakmask); 
  
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), idata.begin(),data.begin());
  //cout << "segmented by peakmask sums:" << endl;
  //printvec(1,NDATA,data);
   
  thrust::reverse(data.begin(), data.end());
  thrust::reverse(peakmask.begin(), peakmask.end());
  thrust::equal_to<Numeric> binary_pred;
  thrust::maximum<Numeric>  binary_max;
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), data.begin(), data.begin(),binary_pred,binary_max);
  
  thrust::reverse(data.begin(), data.end());
}



////////////////////////////////////////////////////////////////////////////////////////////////////
// Test of peak mask
////////////////////////////////////////////////////////////////////////////////////////////////////
int main(void)
{
  const int NDATA=20;
  int values[NDATA] = {	0, 0, 2, 3, 1, 0, 0, 1, 2, 0, 0, 1, 2, 3, 4, 0, 0, 1, 2, 0};
  int m = 10, n = 2;
   
  // transfer to device
  thrust::device_vector<int> 	 idata(values, values + NDATA);
  cout << "testing peak mask and integral" << endl;
  cout << "idata (two vectors one after another):" << endl;
  printvec(m,n,idata);

  thrust::device_vector<Numeric> data(m*n);
  
  /*
  thrust::device_vector<Numeric> peakmask(m*n);

  thrust::transform(idata.begin(), idata.end(), peakmask.begin(), mask01<Numeric>(Integer(0)));
  cout << "peakmask after fill ():" << endl;
  printvec(1,NDATA,peakmask); 
  
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), idata.begin(),data.begin());
  cout << "segmented by peakmask sums:" << endl;
  printvec(1,NDATA,data);
   
  thrust::reverse(data.begin(), data.end());
  thrust::reverse(peakmask.begin(), peakmask.end());
  
  thrust::equal_to<Numeric> binary_pred;
  thrust::maximum<Numeric>  binary_max;
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), data.begin(), data.begin(),binary_pred,binary_max);
  
  thrust::reverse(data.begin(), data.end());
  */
  segmented_peak_sums(m, n, idata, data);
  cout << "data after max(sum) in peakmask segments:" << endl;
  printvec(1,NDATA,data); 
  return 0;
}
