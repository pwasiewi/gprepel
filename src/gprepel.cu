#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
//  gpRepel : An R package for GPU computing
//
//  This program is free software; you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation; version 3 of the License.
//
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program; if not, write to the Free Software
//  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
//
//  Author: Piotr Wąsiewicz pwasiewi@gmail.com
//
////////////////////////////////////////////////////////////////////////////////////////////////////

#include <R.h>
#include "gpRepel.h"

typedef thrust::tuple<Numeric,Numeric> 								Numeric2;
typedef typename thrust::device_vector<Numeric>::iterator         	NumericIterator;
typedef typename thrust::tuple<NumericIterator, NumericIterator>  	NumericIteratorTuple;
typedef typename thrust::zip_iterator<NumericIteratorTuple>       	Numeric2Iterator;
typedef thrust::tuple<Numeric,Numeric,Numeric> 						Numeric3;
typedef typename thrust::tuple<NumericIterator, NumericIterator, NumericIterator>  NumericIteratorTuple3;
typedef typename thrust::zip_iterator<NumericIteratorTuple3>       	Numeric3Iterator;

const int NAVG=80;

////////////////////////////////////////////////////////////////////////////////////////////////////
// VecReorder - the gpu functor implementing the dot product between 3d vectors
//
struct VecReorder : public thrust::binary_function<Numeric2,Numeric2,Numeric>
{
    Numeric w, maxb;
    Numeric ai,bi,av,bv, result;

    __host__ __device__
    VecReorder(Numeric w, Numeric maxb) : w(w), maxb(maxb) {}
    __host__ __device__
        Numeric operator()(const Numeric2& a, const Numeric2& b) const
        {
            Numeric ai=(int) thrust::get<0>(a) % (int) maxb;
            Numeric av=thrust::get<1>(a);
            Numeric bi=(int) thrust::get<0>(b) % (int) maxb;
            Numeric bv=thrust::get<1>(b);
	    	int lastone = (int) maxb*((int) thrust::get<0>(a) / (int) maxb)-1;
	    	if(ai > bi)
            	return thrust::get<0>(a);
	    	else
	    		return lastone;
        }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// minus_and_divide_zip - the gpu functor implementing moving average in a point 
//
struct  minus_and_divide_zip : public thrust::binary_function<Numeric3,Numeric3,Numeric>
{
    Numeric w, maxb;
    Numeric ai,bi,av,bv, result;

    __host__ __device__
    minus_and_divide_zip(Numeric w, Numeric maxb) : w(w), maxb(maxb) {}
    __host__ __device__
    Numeric operator()(const Numeric3& a, const Numeric3& b) const
    {
		Numeric ai=(int) thrust::get<0>(a) % (int) maxb;
		Numeric av=thrust::get<1>(a);
		Numeric bi=(int) thrust::get<0>(b) % (int) maxb;
		Numeric bv=thrust::get<1>(b);
	    int lastone = (int) maxb*((int) thrust::get<0>(a) / (int) maxb);
	    if(ai > bi)
                return (av - bv)/w;
	    else
	    	if((int)w - 1 != (int) ai)
	    		return (thrust::get<2>(b) - thrust::get<1>(b)) / ((int)w - (int)ai -1);
	    	else
	    		return 0;
    }
};


////////////////////////////////////////////////////////////////////////////////////////////////////
// simple_moving_average - GPU function of the simple average with a window w points forward, 
// after a given point; idata, vout - input and output matrices with m (rows) x n (cols) dimensions,
// 
template <typename InputVector, typename OutputVector>
void simple_moving_average(size_t m, size_t n, const InputVector& idata, size_t w, OutputVector& vout)
{
    typedef typename InputVector::value_type T;

    if (idata.size() < w)
        return;
    thrust::device_vector<size_t> output(m*n);
    thrust::device_vector<Numeric> voutput(m*n);
    thrust::device_vector<Numeric> data(m*n);
    thrust::device_vector<Numeric> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end());

    thrust::inclusive_scan(idata.begin(), idata.end(), data.begin());

    Numeric2Iterator first = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin(), data.begin()));
    Numeric2Iterator firstw = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin() + w, data.begin() + w));
    Numeric2Iterator last  = thrust::make_zip_iterator(thrust::make_tuple(vindex.end(),   data.end()));

    thrust::transform(firstw, last, first, output.begin(), VecReorder(w,m));
 
    thrust::gather(output.begin(), output.end(), data.begin(), voutput.begin());

    Numeric3Iterator first3 = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin(), data.begin(), voutput.begin()));
    Numeric3Iterator firstw3 = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin() + w, data.begin() + w, voutput.begin() + w));
    Numeric3Iterator last3  = thrust::make_zip_iterator(thrust::make_tuple(vindex.end(), data.end(), voutput.end()));

    thrust::transform(firstw3, last3, first3, vout.begin(), minus_and_divide_zip(w,m));
    thrust::fill(vout.end()-w,vout.end(),vout[vout.size()-w-1]);
}



////////////////////////////////////////////////////////////////////////////////////////////////////
// gprpostmave - host-gpu function for the gpu simple average function simple_moving_average 
// with a window w points forward, after a given point
// pint, pout - input and output matrices with a (rows) x b (cols) dimensions
//
void gprpostmave(PNumeric pint, PInteger a, PInteger b, PInteger win, PNumeric pout) {

    // window size of the moving average
    size_t w = win[0];
    size_t m = a[0];//row number
    size_t n = b[0];//column number

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);

    simple_moving_average(m,n,gveca, w, gvecb);

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// print an array m x n with vectors in columns
//
template <typename T>
void print(size_t m, size_t n, thrust::device_vector<T>& d_data)
{
    thrust::host_vector<T> h_data = d_data;

    for(size_t i = 0; i < n; i++)
    {
        for(size_t j = 0; j < m; j++)
            std::cout << " " << h_data[j + i * m] << " ";
        std::cout << "\n";
    }
    std::cout << "\n";
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// print an array m x n with vectors in columns as one vector: one after another
////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T>
void printvec(size_t m, size_t n, thrust::device_vector<T>& d_data)
{
    thrust::host_vector<T> h_data = d_data;

    for(size_t i = 0; i < m*n; i++)
        //std::cout << fixed << setw(5) << setprecision(2) << h_data[i] << " ";
        std::cout << " " << h_data[i] << " ";
    std::cout << "\n";
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// gprpremave - host-gpu function for the gpu simple average function simple_moving_average 
// with a window w points back, before a given point
// pint, pout - input and output matrices with a (rows) x b (cols) dimensions
//
void gprpremave(PNumeric pint, PInteger a, PInteger b, PInteger win, PNumeric pout) {

    // window size of the moving average
    size_t w = win[0];
    size_t m = a[0];//row number
    size_t n = b[0];//column number

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);

    thrust::reverse(gveca.begin(), gveca.end());
    simple_moving_average(m,n,gveca, w, gvecb);
    thrust::reverse(gvecb.begin(), gvecb.end());
    thrust::reverse(gveca.begin(), gveca.end());

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// plus_and_divide - a gpu functor of two arguments added and divided by a constant
//
template <typename T>
struct plus_and_divide : public thrust::binary_function<T,T,T>
{
    T w;

    __host__ __device__
    plus_and_divide(T w) : w(w) {}

    __host__ __device__
    T operator()(const T& a, const T& b) const
    {
        return (a + b) / w;
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// double_moving_average - gpu function for the gpu average of two moving averages with windows 
// after and before the given point, joint of two simple_moving_average (one on a reversed copy)
template <typename InputVector, typename OutputVector>
void double_moving_average(size_t m, size_t n, const InputVector& igva, size_t w, OutputVector& gvd)
{
    typedef typename InputVector::value_type T;
    if (igva.size() < w)
        return;

    thrust::device_vector<T> gva(igva.size());
    thrust::device_vector<T> gvb(igva.size());
    thrust::device_vector<T> gvc(igva.size());
    thrust::copy(igva.begin(), igva.end(), gva.begin());

    simple_moving_average(m,n,gva, w, gvb);
    thrust::reverse(gva.begin(), gva.end());
    simple_moving_average(m,n,gva, w, gvc);
    thrust::reverse(gvc.begin(), gvc.end());
    thrust::reverse(gva.begin(), gva.end());
    thrust::transform(gvc.begin(), gvc.end(), gvb.begin(), gvd.begin(), plus_and_divide<T>(T(2)));
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// gprpremave - host-gpu function for the gpu average of two moving averages with windows 
// after and before the given point, joint of gprpremave and gprpostmave
// pint, pout - input and output matrices with a (rows) x b (cols) dimensions
//
void gprmoverage(PNumeric pint, PInteger a, PInteger b, PInteger win, PNumeric pout) {

    // window size of the moving average
    size_t w = win[0];
    size_t m = a[0];//row number
    size_t n = b[0];//column number

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecz(a[0]*b[0]);
    double_moving_average(m,n,gveca,w,gvecz);

    // transfer data back to host
    thrust::copy(gvecz.begin(), gvecz.end(), pout);
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// gprbasavoff - host-gpu function for a wave average with a small window divided by its baseline 
// (double average with a large window)
//
void gprbasavoff(PNumeric pint, PInteger a, PInteger b, PInteger win1, PInteger win2, PNumeric pout) {

    // window size of the moving average
    size_t w1 = win1[0];//smaller window
    size_t w2 = win2[0];//larger window
    size_t m = a[0];//row number
    size_t n = b[0];//column number

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecc(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecd(a[0]*b[0]);

    double_moving_average(m,n,gveca, w1, gvecb);
    double_moving_average(m,n,gveca, w2, gvecc);

    thrust::transform(gvecb.begin(), gvecb.end(), gvecc.begin(), gvecd.begin(), thrust::divides<Numeric>());

    // transfer data back to host
    thrust::copy(gvecd.begin(), gvecd.end(), pout);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// gprbasoroff - host-gpu function for a wave divided by its baseline (double average 
// with a large window)
//
void gprbasoroff(PNumeric pint, PInteger a, PInteger b, PInteger win1, PNumeric pout) {

    // window size of the moving average
    size_t w1 = win1[0];//smaller window

    size_t m = a[0];//row number
    size_t n = b[0];//column number

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);

    double_moving_average(m,n,gveca, w1, gvecb);

    thrust::transform(gveca.begin(), gveca.end(), gvecb.begin(), gveca.begin(), thrust::divides<Numeric>());

    // transfer data back to host
    thrust::copy(gveca.begin(), gveca.end(), pout);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// gprdiff - host-gpu function for shifted by the w window one wave copies substraction
//
void gprdiff(PNumeric pint, PInteger a, PInteger b, PInteger win1, PNumeric pout) {
    size_t w = win1[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::transform(gveca.begin()+w, gveca.end(), gveca.begin(), gvecb.begin(), thrust::minus<Numeric>());

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// gprdiffrev - host-gpu function for shifted by the w window one wave reversed copies substraction
//
void gprdiffrev(PNumeric pint, PInteger a, PInteger b, PInteger win1, PNumeric pout) {
    size_t w = win1[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::reverse(gveca.begin(), gveca.end());
    thrust::transform(gveca.begin()+w, gveca.end(), gveca.begin(), gvecb.begin(), thrust::minus<Numeric>());
    thrust::reverse(gvecb.begin(), gvecb.end());

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// oneup - the gpu functor of constant value substraction, where negative values are set to 0
//
template <typename T>
struct oneup : public thrust::unary_function<T,T>
{
    T w;
	__host__ __device__
    oneup(T w) : w(w) {}

	__host__ __device__
    T operator()(const T& a) const
    {
    	if(a <= w)
    		return 0;
    	else
    		return a-w;
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// gprup - host-gpu function for one wave and a horizontal line substraction 
// (negative values are set to 0)
// 
void gprup(PNumeric pint, PInteger a, PInteger b, PNumeric win1, PNumeric pout) {
	Numeric w1 = win1[0];

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::transform(gveca.begin(), gveca.end(), gvecb.begin(), oneup<Numeric>(Numeric(w1)));

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// onedown - the gpu functor from constant value the wave substraction, 
// where negative values are set to 0
//
template <typename T>
struct onedown : public thrust::unary_function<T,T>
{
    T w;
	__host__ __device__
    onedown(T w) : w(w) {}

    __host__ __device__
    T operator()(const T& a) const
    {
    	if(a >= w)
    		return 0;
    	else
    		return w-a;
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// gprdown - host-gpu function for a horizontal line and one wave substraction 
// (negative values are set to 0)
// 
void gprdown(PNumeric pint, PInteger a, PInteger b, PNumeric win1, PNumeric pout) {
	Numeric w1 = win1[0];

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::transform(gveca.begin(), gveca.end(), gvecb.begin(), onedown<Numeric>(Numeric(w1)));

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// minus_by - the gpu functor of the wave and the constant value substraction square 
//
template<typename T>
struct  minus_by: public thrust::unary_function<T,T>
{
    T w;
	__host__ __device__
    minus_by(T w) : w(w) {}

   __host__ __device__
   T operator()(const T &x) const
   {
    return (x - w)*(x - w);
   }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// gprsdall - host-gpu function for a squared global average substraction sum 
// divided by a number of vectors   
// sqrt((x - globalavg)^2)/N
// 
void gprsdall(PNumeric pint, PInteger a, PInteger b, PNumeric pout) {

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    Numeric sumall = thrust::reduce(gveca.begin(), gveca.end())/(a[0]*b[0]);
    Numeric result = thrust::transform_reduce(gveca.begin(), gveca.end(),
                                            minus_by<Numeric>(Numeric(sumall)),
                                            0,
                                            thrust::plus<Numeric>());
    pout[0] = sqrt(result/(a[0]*b[0]));
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// gpravgall - host-gpu function for a global average of all vectors  
// 
void gpravgall(PNumeric pint, PInteger a, PInteger b, PNumeric pout) {

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

	pout[0] = thrust::reduce(gveca.begin(), gveca.end())/(a[0]*b[0]);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// kindcreate - the gpu functor of integer division by a constant m
//
struct kindcreate : public thrust::unary_function<Numeric,Numeric>
{
    Integer m, n;

    __host__ __device__
    kindcreate(Integer m, Integer n) : m(m), n(n) {}
    __host__ __device__
    Numeric operator()(const Numeric& a) const
    {
        //Integer ai=(int) ((int) a) % m;
        Integer av=(int) ((int) a) / m;
        //if(a > 0)
        return (int) av;
   }
};



////////////////////////////////////////////////////////////////////////////////////////////////////
// gprmovemax - host-gpu function for finding a global maximum horizontal line for each vector
//
void gprmovemax(PNumeric pint, PInteger a, PInteger b, PInteger win1, PNumeric pout) {

    // window size of the moving average
    //int w = win1[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Integer> vindex(a[0]*b[0]);
    thrust::sequence(vindex.begin(),vindex.end(),0);
    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecc(a[0]*b[0]);
    thrust::transform(vindex.begin(), vindex.end(), gvecb.begin(), kindcreate(Integer(a[0]),Integer(b[0])));

    thrust::equal_to<Numeric> binary_pred;
    thrust::maximum<Numeric>   binary_op;
    thrust::inclusive_scan_by_key(gvecb.begin(), gvecb.end(), gveca.begin(), gvecc.begin(),binary_pred,binary_op);
    thrust::reverse(gvecc.begin(), gvecc.end());
    thrust::inclusive_scan_by_key(gvecb.begin(), gvecb.end(), gvecc.begin(), gveca.begin(),binary_pred,binary_op);
    thrust::reverse(gveca.begin(), gveca.end());

    // transfer data back to host
    thrust::copy(gveca.begin(), gveca.end(), pout);
    //thrust::copy(vindex.begin(), vindex.end(), pout);
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// meanorig - gpu function for for the divided by m wave mean without baseline  
//
template <typename InputVector, typename OutputVector>
void meanorig(size_t m, size_t n, const InputVector& gveca, OutputVector& vout)
{
    thrust::device_vector<Numeric> orgbasoff(m*n);

    thrust::device_vector<Integer> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end(),0);
    thrust::transform(vindex.begin(), vindex.end(), vindex.begin(), kindcreate(Integer(m),Integer(n)));
    thrust::device_vector<Numeric> gvecb(m*n);
//    thrust::device_vector<Numeric> gvecc(m*n);
    thrust::device_vector<Numeric> gvecd(m*n);

    double_moving_average(m,n,gveca, 150, gvecb);
    thrust::transform(gveca.begin(), gveca.end(), gvecb.begin(), orgbasoff.begin(), thrust::divides<Numeric>());
    thrust::transform(orgbasoff.begin(), orgbasoff.end(), orgbasoff.begin(), oneup<Numeric>(Numeric(1)));

    thrust::equal_to<Numeric> binary_pred;
    thrust::maximum<Numeric>  binary_max;
/*
    thrust::device_vector<Numeric> avgbasoff(a[0]*b[0]);
    double_moving_average(a[0],b[0],gveca, 150, gvecb);
    double_moving_average(a[0],b[0],gveca,  80, gvecc);
    thrust::transform(gvecc.begin(), gvecc.end(), gvecb.begin(), avgbasoff.begin(), thrust::divides<Numeric>());
    thrust::transform(avgbasoff.begin(), avgbasoff.end(), avgbasoff.begin(), oneup<Numeric>(Numeric(1)));


    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), avgbasoff.begin(), gvecb.begin(),binary_pred,binary_max);
    thrust::reverse(gvecb.begin(), gvecb.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), gvecb.begin(), gvecc.begin(),binary_pred,binary_max);
    thrust::reverse(gvecc.begin(), gvecc.end());
*/
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoff.begin(), gvecb.begin(),binary_pred,thrust::plus<Numeric>());
    thrust::reverse(gvecb.begin(), gvecb.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), gvecb.begin(), gvecd.begin(),binary_pred,binary_max);
    thrust::reverse(gvecd.begin(), gvecd.end());
    thrust::fill(gvecb.begin(),gvecb.end(),m);
    thrust::transform(gvecd.begin(), gvecd.end(), gvecb.begin(), vout.begin(), thrust::divides<Numeric>());
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// gprmeanmax - host-gpu function for the divided by m wave mean without baseline  
//
void gprmeanmax(PNumeric pint, PInteger a, PInteger b, PInteger win1, PNumeric pout) {

    // window size of the moving average
    //int w = win1[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());
    thrust::device_vector<Numeric> meanvec(a[0]*b[0]);

    meanorig(a[0], b[0], gveca, meanvec);
    // transfer data back to host
    thrust::copy(meanvec.begin(), meanvec.end(), pout);
    //thrust::copy(vindex.begin(), vindex.end(), pout);
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// compare_zip - the gpu functor 
//
struct  compare_zip : public thrust::binary_function<Numeric2,Numeric2,Numeric>
{
     Numeric va,vb, da, db;

    __host__ __device__
    Numeric operator()(const Numeric2& a, const Numeric2& b) const
    {
		Numeric va=thrust::get<0>(a);
		Numeric da=thrust::get<1>(a);
		Numeric vb=thrust::get<0>(b);
		Numeric db=thrust::get<1>(b);
	    if(va > 0){
	    	if(da > 0 && db <0){
	    		return 1;
	    	}
	    	else
	    		return 0;
	    }
	    else
	    	return 0;
     }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// zipup - the gpu functor 
//
struct  zipup : public thrust::unary_function<Numeric2,Numeric>
{
     Numeric va,da;

    __host__ __device__
    Numeric operator()(const Numeric2& a) const
    {
		Numeric va=thrust::get<0>(a);
		Numeric da=thrust::get<1>(a);
    	if(va <= da)
    		return 0;
    	else
    		return va-da;
     }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// find_maxpeaks - gpu function for 
//
template <typename InputVector, typename OutputVector>
void find_maxpeaks(size_t m, size_t n, const InputVector& gveca, const Numeric& w1, const Numeric& w2, const Numeric& up, OutputVector& vout, OutputVector& avgbasoff, OutputVector& orgbasoff)
{
	//thrust::device_vector<Numeric> orgbasoff(m*n);

    thrust::device_vector<Integer> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end(),0);
    thrust::transform(vindex.begin(), vindex.end(), vindex.begin(), kindcreate(Integer(m),Integer(n)));
    thrust::device_vector<Numeric> gvecb(m*n);
    thrust::device_vector<Numeric> gvecc(m*n);
    thrust::device_vector<Numeric> gvecd(m*n);

    double_moving_average(m,n,gveca, w2, gvecb);
    thrust::transform(gveca.begin(), gveca.end(), gvecb.begin(), orgbasoff.begin(), thrust::divides<Numeric>());

    if(up > 0)
    	thrust::transform(orgbasoff.begin(), orgbasoff.end(), orgbasoff.begin(), oneup<Numeric>(Numeric(1)));
    else
    	thrust::transform(orgbasoff.begin(), orgbasoff.end(), orgbasoff.begin(), onedown<Numeric>(Numeric(1)));

    thrust::equal_to<Numeric> binary_pred;
    thrust::maximum<Numeric>  binary_max;

    //thrust::device_vector<Numeric> avgbasoff(m*n);
    // up and down peaks from orig and avg divided by baseline
    double_moving_average(m,n,gveca, w2, gvecb);
    double_moving_average(m,n,gveca, w1, gvecc);
    thrust::transform(gvecc.begin(), gvecc.end(), gvecb.begin(), avgbasoff.begin(), thrust::divides<Numeric>());
    if(up > 0)
    	thrust::transform(avgbasoff.begin(), avgbasoff.end(), avgbasoff.begin(), oneup<Numeric>(Numeric(1)));
    else
    	thrust::transform(avgbasoff.begin(), avgbasoff.end(), avgbasoff.begin(), onedown<Numeric>(Numeric(1)));

    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoff.begin(), gvecb.begin(),binary_pred,thrust::plus<Numeric>());
    thrust::reverse(gvecb.begin(), gvecb.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), gvecb.begin(), gvecd.begin(),binary_pred,binary_max);
    thrust::reverse(gvecd.begin(), gvecd.end());
    thrust::fill(gvecb.begin(),gvecb.end(),m);
    thrust::transform(gvecd.begin(), gvecd.end(), gvecb.begin(), gvecd.begin(), thrust::divides<Numeric>());

    Numeric2Iterator first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoff.begin(), gvecd.begin()));
    Numeric2Iterator last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoff.end(),   gvecd.end()));

    thrust::transform(first, last, gvecb.begin(), zipup());

	//substraction by 1
    thrust::transform(gvecb.begin()+1, gvecb.end(), gvecb.begin(), gvecc.begin(), thrust::minus<Numeric>());
    double_moving_average(m,n,gvecc, 20, gvecd);

    Numeric2Iterator first0 = thrust::make_zip_iterator(thrust::make_tuple(gvecb.begin(), gvecd.begin()));
    Numeric2Iterator first1 = thrust::make_zip_iterator(thrust::make_tuple(gvecb.begin() + 1, gvecd.begin() + 1));
    Numeric2Iterator last0  = thrust::make_zip_iterator(thrust::make_tuple(gvecb.end(),  gvecd.end()));

    thrust::transform(first0, last0, first1, vout.begin(), compare_zip());

}



////////////////////////////////////////////////////////////////////////////////////////////////////
// gprpeakmask - the host-gpu function for 
//
void gprpeakmask(PNumeric pint, PInteger a, PInteger b, PNumeric win1, PNumeric win2, PNumeric up, PNumeric pout) {
    size_t m = a[0];//row number
    size_t n = b[0];//column number
    int w1 = win1[0];//difference window
    int w2 = win2[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());
    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecc(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecd(a[0]*b[0]);

    find_maxpeaks(m,n,gveca,w1,w2,up[0],gvecb,gvecc,gvecd);
    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
    thrust::copy(gvecc.begin(), gvecc.end(), pout+m*n+1);
    thrust::copy(gvecd.begin(), gvecd.end(), pout+2*m*n+1);
}




////////////////////////////////////////////////////////////////////////////////////////////////////
// upmask - the gpu functor 
//
struct  upmask : public thrust::unary_function<Numeric,Numeric>
{
	Numeric out;
	__host__ __device__
    upmask(Numeric out) : out(out) {}
    __host__ __device__
    Numeric operator()(const Numeric& a)
    {
    	if(a > 0){
    		return out;
    	}
    	else
    	{
    		out=out+1;
    		return 0;
    	}
     }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// find_doublemaxpeaks - a gpu function for 
//
template <typename InputVector, typename OutputVector>
void find_doublemaxpeaks(size_t m, size_t n, const InputVector& idata, const Numeric& w1, const Numeric& w2, const Numeric& w3, OutputVector& pout, OutputVector& avgbasoffp, OutputVector& dout, OutputVector& avgbasoffd)
{
	thrust::device_vector<Numeric> orgbasoffp(m*n);
	thrust::device_vector<Numeric> orgbasoffd(m*n);

    thrust::device_vector<Numeric> data(m*n);
    thrust::device_vector<Numeric> data2(m*n);

    double_moving_average(m,n,idata, w2, data2);
    thrust::transform(idata.begin(), idata.end(), data2.begin(), data.begin(), thrust::divides<Numeric>());

    //original peaks up and down
    thrust::transform(data.begin(), data.end(), orgbasoffp.begin(), oneup<Numeric>(Numeric(1)));
    thrust::transform(data.begin(), data.end(), orgbasoffd.begin(), onedown<Numeric>(Numeric(1)));

    thrust::equal_to<Numeric> binary_pred;
    thrust::maximum<Numeric>  binary_max;
 
    double_moving_average(m,n,idata, w1, data);
    thrust::transform(data.begin(), data.end(), data2.begin(), data.begin(), thrust::divides<Numeric>());
	//average peaks up and down
    thrust::transform(data.begin(), data.end(), avgbasoffp.begin(), oneup<Numeric>(Numeric(1)));
    thrust::transform(data.begin(), data.end(), avgbasoffd.begin(), onedown<Numeric>(Numeric(1)));

	//the same number within each vector
    thrust::device_vector<Integer> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end(),0);
    thrust::transform(vindex.begin(), vindex.end(), vindex.begin(), kindcreate(Integer(m),Integer(n)));

    //orgbasoffp, avgbasoffp
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoffp.begin(), data.begin(), binary_pred,thrust::plus<Numeric>());
    thrust::reverse(data.begin(), data.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), data.begin(), data2.begin(),binary_pred,binary_max);
    thrust::reverse(data2.begin(), data2.end());
    thrust::fill(data.begin(),data.end(),m/(NAVG/w1));
    thrust::transform(data2.begin(), data2.end(), data.begin(), data2.begin(), thrust::divides<Numeric>());

    Numeric2Iterator first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffp.begin(), data2.begin()));
    Numeric2Iterator last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffp.end(),   data2.end()));

    thrust::transform(first, last, data.begin(), zipup());

    thrust::transform(data.begin()+1, data.end(), data.begin(), pout.begin(), thrust::minus<Numeric>());
    double_moving_average(m,n,pout, w3, data2);

    Numeric2Iterator first0 = thrust::make_zip_iterator(thrust::make_tuple(data.begin(), data2.begin()));
    Numeric2Iterator first1 = thrust::make_zip_iterator(thrust::make_tuple(data.begin() + 1, data2.begin() + 1));
    Numeric2Iterator last0  = thrust::make_zip_iterator(thrust::make_tuple(data.end(),  data2.end()));

    thrust::transform(first0, last0, first1, pout.begin(), compare_zip());

    //orgbasoffd, avgbasoffd
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoffd.begin(), data.begin(), binary_pred,thrust::plus<Numeric>());
    thrust::reverse(data.begin(), data.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), data.begin(), data2.begin(),binary_pred,binary_max);
    thrust::reverse(data2.begin(), data2.end());
    thrust::fill(data.begin(),data.end(),m/(NAVG/w1));
    thrust::transform(data2.begin(), data2.end(), data.begin(), data2.begin(), thrust::divides<Numeric>());

    first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffd.begin(), data2.begin()));
    last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffd.end(),   data2.end()));

    thrust::transform(first, last, data.begin(), zipup());

    thrust::transform(data.begin()+1, data.end(), data.begin(), dout.begin(), thrust::minus<Numeric>());
    double_moving_average(m,n,dout, w3, data2);

    first0 = thrust::make_zip_iterator(thrust::make_tuple(data.begin(), data2.begin()));
    first1 = thrust::make_zip_iterator(thrust::make_tuple(data.begin() + 1, data2.begin() + 1));
    last0  = thrust::make_zip_iterator(thrust::make_tuple(data.end(),  data2.end()));
    
    thrust::transform(first0, last0, first1, dout.begin(), compare_zip());
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// gprpeak2mask - the host-gpu function for 
//
void gprpeak2mask(PNumeric pint, PInteger a, PInteger b, PNumeric win1, PNumeric win2, PNumeric win3, PNumeric pout) {
    size_t m = a[0];//row number
    size_t n = b[0];//column number
    int w1 = win1[0];//window
    int w2 = win2[0];//baseline window
    int w3 = win3[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(pint,pint+a[0]*b[0]);
    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecc(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecd(a[0]*b[0]);
    thrust::device_vector<Numeric> gvece(a[0]*b[0]);

    find_doublemaxpeaks(m,n,gveca,w1,w2,w3,gvecb,gvecc,gvecd,gvece);
    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
    thrust::copy(gvecc.begin(), gvecc.end(), pout+m*n+1);
    thrust::copy(gvecd.begin(), gvecd.end(), pout+2*m*n+1);
    thrust::copy(gvece.begin(), gvece.end(), pout+3*m*n+1);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// avg find peaks 
// 
template <typename InputVector, typename OutputVector>
void avg_find_peaks(size_t m, size_t n, const Numeric& w1, const Numeric& w2, const Numeric& w3, 
InputVector& orgbasoff, InputVector& avgbasoff,OutputVector& out)
{
  typedef typename InputVector::value_type T;
  thrust::equal_to<Numeric> binary_pred;
  thrust::maximum<Numeric>  binary_max;

  thrust::device_vector<Numeric> data(m*n);
  thrust::device_vector<Numeric> data2(m*n);
  
  //the same number within each vector
  thrust::device_vector<Integer> vindex(m*n);
  thrust::sequence(vindex.begin(),vindex.end(),0);
  thrust::transform(vindex.begin(), vindex.end(), vindex.begin(), kindcreate(Integer(m),Integer(n)));
  
  thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoff.begin(), data.begin(), binary_pred,thrust::plus<Numeric>());
  thrust::reverse(data.begin(), data.end());
  thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), data.begin(), data2.begin(),binary_pred,binary_max);
  thrust::reverse(data2.begin(), data2.end());
  thrust::fill(data.begin(),data.end(),m/(NAVG/w1));
  thrust::transform(data2.begin(), data2.end(), data.begin(), data2.begin(), thrust::divides<Numeric>());
  
  	//cout << "peaks divided by average:" << endl;
  	//printvec(1,NDATA,data2);
  	 
  Numeric2Iterator first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoff.begin(), data2.begin()));
  Numeric2Iterator last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoff.end(),   data2.end()));

  thrust::transform(first, last, data.begin(), zipup());

  	//cout << "peaks zipup:" << endl;
  	//printvec(1,NDATA,data);

  thrust::transform(data.begin()+1, data.end(), data.begin(), out.begin(), thrust::minus<Numeric>());
  double_moving_average(m,n,out, w3, data2);

  Numeric2Iterator first0 = thrust::make_zip_iterator(thrust::make_tuple(data.begin(), data2.begin()));
  Numeric2Iterator first1 = thrust::make_zip_iterator(thrust::make_tuple(data.begin() + 1, data2.begin() + 1));
  Numeric2Iterator last0  = thrust::make_zip_iterator(thrust::make_tuple(data.end(),  data2.end()));

  thrust::transform(first0, last0, first1, out.begin(), compare_zip());
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// segregate peaks up and down
// 
template <typename InputVector, typename OutputVector>
void create_peaks(InputVector& data, InputVector& base, OutputVector& wavbasoffp, OutputVector& wavbasoffd)
{
	typedef typename InputVector::value_type T;
	thrust::device_vector<Numeric> data2(data.begin(),data.end());
   	thrust::transform(data.begin(), data.end(), base.begin(), data2.begin(), thrust::divides<Numeric>());
  	thrust::transform(data2.begin(), data2.end(), wavbasoffp.begin(), oneup<Numeric>(Numeric(1)));
  	thrust::transform(data2.begin(), data2.end(), wavbasoffd.begin(), onedown<Numeric>(Numeric(1)));
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// mask01 - the gpu functor of making mask, where values greater than w are set to 1
//
template <typename T>
struct mask01 : public thrust::unary_function<T,T>
{
    T w;
	__host__ __device__
    mask01(T w) : w(w) {}

	__host__ __device__
    T operator()(const T& a) const
    {
    	if(a > w)
    		return 1;
    	else
    		return 0;
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// tone - the gpu functor of making 1, where values are equal to 1
//
template <typename T>
struct tone : public thrust::unary_function<T,T>
{
    T w;
	__host__ __device__
    tone(T w) : w(w) {}

	__host__ __device__
    T operator()(const T& a) const
    {
    	if(a > w)
    		return a;
    	else
    		return 1;
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// segmented by peaks sums 
// 
template <typename InputVector, typename OutputVector>
void segmented_peak_sums(size_t m, size_t n, const InputVector& idata, OutputVector& data)
{
  typedef typename InputVector::value_type T;

  thrust::device_vector<Numeric> peakmask(m*n);

  thrust::transform(idata.begin(), idata.end(), peakmask.begin(), mask01<Numeric>(Integer(0)));
  //thrust::device_vector<Numeric> peakbase(peakmask.begin(), peakmask.end());
  //thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), peakbase.begin(),peakbase.begin());
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), idata.begin(),data.begin());
  //thrust::reverse(peakbase.begin(), peakbase.end());
  thrust::reverse(data.begin(), data.end());
  thrust::reverse(peakmask.begin(), peakmask.end());
  thrust::equal_to<Numeric> binary_pred;
  thrust::maximum<Numeric>  binary_max;
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), data.begin(), data.begin(),binary_pred,binary_max);
  //thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), peakbase.begin(), peakbase.begin(),binary_pred,binary_max);
  //thrust::reverse(peakbase.begin(), peakbase.end());
  thrust::reverse(data.begin(), data.end());
  //thrust::transform(peakbase.begin(), peakbase.end(), peakbase.begin(), tone<Numeric>(Integer(0)));
  //cout << "data of peakbase:" << endl;
  //printvec(1,NDATA,peakbase); 
  //thrust::transform(data.begin(), data.end(), peakbase.begin(), data.begin(), thrust::divides<Numeric>());

}

////////////////////////////////////////////////////////////////////////////////////////////////////
// find_doublemaxpeaks_opt - a gpu function for 
//
template <typename InputVector, typename OutputVector>
void find_doublemaxpeaks_opt(size_t m, size_t n, const InputVector& idata, const Numeric& w1, const Numeric& w2, const Numeric& w3, 
OutputVector& pout, OutputVector& avgbasoffp, OutputVector& pintegralp, OutputVector& dout, OutputVector& avgbasoffd, OutputVector& pintegrald)
{
	thrust::device_vector<Numeric> orgbasoffp(m*n);
	thrust::device_vector<Numeric> orgbasoffd(m*n);

    thrust::device_vector<Numeric> data(m*n);
    thrust::device_vector<Numeric> data2(m*n);

    double_moving_average(m,n,idata, w2, data2);
    thrust::transform(idata.begin(), idata.end(), data2.begin(), data.begin(), thrust::divides<Numeric>());

    //original peaks up and down
    thrust::transform(data.begin(), data.end(), orgbasoffp.begin(), oneup<Numeric>(Numeric(1)));
    thrust::transform(data.begin(), data.end(), orgbasoffd.begin(), onedown<Numeric>(Numeric(1)));

    thrust::equal_to<Numeric> binary_pred;
    thrust::maximum<Numeric>  binary_max;
 
    double_moving_average(m,n,idata, w1, data);
    thrust::transform(data.begin(), data.end(), data2.begin(), data.begin(), thrust::divides<Numeric>());
	//average peaks up and down
    thrust::transform(data.begin(), data.end(), avgbasoffp.begin(), oneup<Numeric>(Numeric(1)));
    thrust::transform(data.begin(), data.end(), avgbasoffd.begin(), onedown<Numeric>(Numeric(1)));

	//the same number within each vector
    thrust::device_vector<Integer> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end(),0);
    thrust::transform(vindex.begin(), vindex.end(), vindex.begin(), kindcreate(Integer(m),Integer(n)));

    //orgbasoffp, avgbasoffp
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoffp.begin(), data.begin(), binary_pred,thrust::plus<Numeric>());
    thrust::reverse(data.begin(), data.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), data.begin(), data2.begin(),binary_pred,binary_max);
    thrust::reverse(data2.begin(), data2.end());
    thrust::fill(data.begin(),data.end(),m/(NAVG/w1));
    thrust::transform(data2.begin(), data2.end(), data.begin(), data2.begin(), thrust::divides<Numeric>());

    Numeric2Iterator first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffp.begin(), data2.begin()));
    Numeric2Iterator last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffp.end(),   data2.end()));

    thrust::transform(first, last, data.begin(), zipup());

    thrust::transform(data.begin()+1, data.end(), data.begin(), pout.begin(), thrust::minus<Numeric>());
    double_moving_average(m,n,pout, w3, data2);

    Numeric2Iterator first0 = thrust::make_zip_iterator(thrust::make_tuple(data.begin(), data2.begin()));
    Numeric2Iterator first1 = thrust::make_zip_iterator(thrust::make_tuple(data.begin() + 1, data2.begin() + 1));
    Numeric2Iterator last0  = thrust::make_zip_iterator(thrust::make_tuple(data.end(),  data2.end()));

    thrust::transform(first0, last0, first1, pout.begin(), compare_zip());

    //orgbasoffd, avgbasoffd
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoffd.begin(), data.begin(), binary_pred,thrust::plus<Numeric>());
    thrust::reverse(data.begin(), data.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), data.begin(), data2.begin(),binary_pred,binary_max);
    thrust::reverse(data2.begin(), data2.end());
    thrust::fill(data.begin(),data.end(),m/(NAVG/w1));
    thrust::transform(data2.begin(), data2.end(), data.begin(), data2.begin(), thrust::divides<Numeric>());

    first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffd.begin(), data2.begin()));
    last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffd.end(),   data2.end()));

    thrust::transform(first, last, data.begin(), zipup());

    thrust::transform(data.begin()+1, data.end(), data.begin(), dout.begin(), thrust::minus<Numeric>());
    double_moving_average(m,n,dout, w3, data2);

    first0 = thrust::make_zip_iterator(thrust::make_tuple(data.begin(), data2.begin()));
    first1 = thrust::make_zip_iterator(thrust::make_tuple(data.begin() + 1, data2.begin() + 1));
    last0  = thrust::make_zip_iterator(thrust::make_tuple(data.end(),  data2.end()));
    
    thrust::transform(first0, last0, first1, dout.begin(), compare_zip());
    segmented_peak_sums(m, n, avgbasoffp, pintegralp);
    segmented_peak_sums(m, n, avgbasoffd, pintegrald);
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// gprpeak2maskopt - the host-gpu function for 
//
void gprpeak2maskopt(PNumeric pint, PInteger a, PInteger b, PNumeric win1, PNumeric win2, PNumeric win3, PNumeric pout) {
    size_t m = a[0];//row number
    size_t n = b[0];//column number
    int w1 = win1[0];//window
    int w2 = win2[0];//baseline window
    int w3 = win3[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(pint,pint+a[0]*b[0]);
    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecc(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecd(a[0]*b[0]);
    thrust::device_vector<Numeric> gvece(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecf(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecg(a[0]*b[0]);

    find_doublemaxpeaks_opt(m,n,gveca,w1,w2,w3,gvecb,gvecc,gvecd,gvece,gvecf,gvecg);
    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
    thrust::copy(gvecc.begin(), gvecc.end(), pout+m*n+1);
    thrust::copy(gvecd.begin(), gvecd.end(), pout+2*m*n+1);
    thrust::copy(gvece.begin(), gvece.end(), pout+3*m*n+1);
    thrust::copy(gvecf.begin(), gvecf.end(), pout+4*m*n+1);
    thrust::copy(gvecg.begin(), gvecg.end(), pout+5*m*n+1);
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// segmented by peaks multiplied by lambdadiff sums 
// 
template <typename InputVector, typename OutputVector>
void lambda_peak_sums(size_t m, size_t n, const InputVector& ldiff, const InputVector& idata, OutputVector& data)
{
  typedef typename InputVector::value_type T;

  thrust::device_vector<Numeric> peakmask(m*n);
  thrust::device_vector<Numeric> sdata(m*n);

  thrust::transform(idata.begin(), idata.end(), peakmask.begin(), mask01<Numeric>(Integer(0)));
  thrust::transform(ldiff.begin(), ldiff.end(), idata.begin(), sdata.begin(), thrust::multiplies<Numeric>());  
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), sdata.begin(),data.begin());
  thrust::reverse(data.begin(), data.end());
  thrust::reverse(peakmask.begin(), peakmask.end());
  thrust::equal_to<Numeric> binary_pred;
  thrust::maximum<Numeric>  binary_max;
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), data.begin(), data.begin(),binary_pred,binary_max);
  thrust::reverse(data.begin(), data.end());
  
  //thrust::transform(peakbase.begin(), peakbase.end(), peakbase.begin(), tone<Numeric>(Integer(0)));
  //cout << "data of peakbase:" << endl;
  //printvec(1,NDATA,peakbase); 
  //thrust::transform(data.begin(), data.end(), peakbase.begin(), data.begin(), thrust::divides<Numeric>());

}

////////////////////////////////////////////////////////////////////////////////////////////////////
// find_doublemaxpeaks_lambda - a gpu function for 
//
template <typename InputVector, typename OutputVector>
void find_doublemaxpeaks_lambda(size_t m, size_t n, const InputVector& lambda, const InputVector& idata, const Numeric& w1, const Numeric& w2, const Numeric& w3, 
OutputVector& pout, OutputVector& avgbasoffp, OutputVector& pintegralp, OutputVector& dout, OutputVector& avgbasoffd, OutputVector& pintegrald)
{
	thrust::device_vector<Numeric> ldiff(m*n);
	thrust::transform(lambda.begin()+1, lambda.end(), lambda.begin(), ldiff.begin(), thrust::minus<Numeric>());

	thrust::device_vector<Numeric> orgbasoffp(m*n);
	thrust::device_vector<Numeric> orgbasoffd(m*n);

    thrust::device_vector<Numeric> data(m*n);
    thrust::device_vector<Numeric> data2(m*n);

    double_moving_average(m,n,idata, w2, data2);
    thrust::transform(idata.begin(), idata.end(), data2.begin(), data.begin(), thrust::divides<Numeric>());

    //original peaks up and down
    thrust::transform(data.begin(), data.end(), orgbasoffp.begin(), oneup<Numeric>(Numeric(1)));
    thrust::transform(data.begin(), data.end(), orgbasoffd.begin(), onedown<Numeric>(Numeric(1)));

    thrust::equal_to<Numeric> binary_pred;
    thrust::maximum<Numeric>  binary_max;
 
    double_moving_average(m,n,idata, w1, data);
    thrust::transform(data.begin(), data.end(), data2.begin(), data.begin(), thrust::divides<Numeric>());
	//average peaks up and down
    thrust::transform(data.begin(), data.end(), avgbasoffp.begin(), oneup<Numeric>(Numeric(1)));
    thrust::transform(data.begin(), data.end(), avgbasoffd.begin(), onedown<Numeric>(Numeric(1)));

	//the same number within each vector
    thrust::device_vector<Integer> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end(),0);
    thrust::transform(vindex.begin(), vindex.end(), vindex.begin(), kindcreate(Integer(m),Integer(n)));

    //orgbasoffp, avgbasoffp
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoffp.begin(), data.begin(), binary_pred,thrust::plus<Numeric>());
    thrust::reverse(data.begin(), data.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), data.begin(), data2.begin(),binary_pred,binary_max);
    thrust::reverse(data2.begin(), data2.end());
    thrust::fill(data.begin(),data.end(),m/(NAVG/w1));
    thrust::transform(data2.begin(), data2.end(), data.begin(), data2.begin(), thrust::divides<Numeric>());

    Numeric2Iterator first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffp.begin(), data2.begin()));
    Numeric2Iterator last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffp.end(),   data2.end()));

    thrust::transform(first, last, data.begin(), zipup());

    thrust::transform(data.begin()+1, data.end(), data.begin(), pout.begin(), thrust::minus<Numeric>());
    double_moving_average(m,n,pout, w3, data2);

    Numeric2Iterator first0 = thrust::make_zip_iterator(thrust::make_tuple(data.begin(), data2.begin()));
    Numeric2Iterator first1 = thrust::make_zip_iterator(thrust::make_tuple(data.begin() + 1, data2.begin() + 1));
    Numeric2Iterator last0  = thrust::make_zip_iterator(thrust::make_tuple(data.end(),  data2.end()));

    thrust::transform(first0, last0, first1, pout.begin(), compare_zip());

    //orgbasoffd, avgbasoffd
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoffd.begin(), data.begin(), binary_pred,thrust::plus<Numeric>());
    thrust::reverse(data.begin(), data.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), data.begin(), data2.begin(),binary_pred,binary_max);
    thrust::reverse(data2.begin(), data2.end());
    thrust::fill(data.begin(),data.end(),m/(NAVG/w1));
    thrust::transform(data2.begin(), data2.end(), data.begin(), data2.begin(), thrust::divides<Numeric>());

    first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffd.begin(), data2.begin()));
    last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffd.end(),   data2.end()));

    thrust::transform(first, last, data.begin(), zipup());

    thrust::transform(data.begin()+1, data.end(), data.begin(), dout.begin(), thrust::minus<Numeric>());
    double_moving_average(m,n,dout, w3, data2);

    first0 = thrust::make_zip_iterator(thrust::make_tuple(data.begin(), data2.begin()));
    first1 = thrust::make_zip_iterator(thrust::make_tuple(data.begin() + 1, data2.begin() + 1));
    last0  = thrust::make_zip_iterator(thrust::make_tuple(data.end(),  data2.end()));
    
    thrust::transform(first0, last0, first1, dout.begin(), compare_zip());
    lambda_peak_sums(m, n, ldiff, avgbasoffp, pintegralp);
    lambda_peak_sums(m, n, ldiff, avgbasoffd, pintegrald);
}



////////////////////////////////////////////////////////////////////////////////////////////////////
// gprpeaklambda2mask - the host-gpu function for 
//
void gprpeaklambda2mask(PNumeric lint, PNumeric pint, PInteger a, PInteger b, PNumeric win1, PNumeric win2, PNumeric win3, PNumeric pout) {
    size_t m = a[0];//row number
    size_t n = b[0];//column number
    int w1 = win1[0];//window
    int w2 = win2[0];//baseline window
    int w3 = win3[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> lambda(lint,lint+a[0]*b[0]);
    thrust::device_vector<Numeric> gveca(pint,pint+a[0]*b[0]);
    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecc(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecd(a[0]*b[0]);
    thrust::device_vector<Numeric> gvece(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecf(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecg(a[0]*b[0]);

    find_doublemaxpeaks_lambda(m,n,lambda,gveca,w1,w2,w3,gvecb,gvecc,gvecd,gvece,gvecf,gvecg);
    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
    thrust::copy(gvecc.begin(), gvecc.end(), pout+m*n+1);
    thrust::copy(gvecd.begin(), gvecd.end(), pout+2*m*n+1);
    thrust::copy(gvece.begin(), gvece.end(), pout+3*m*n+1);
    thrust::copy(gvecf.begin(), gvecf.end(), pout+4*m*n+1);
    thrust::copy(gvecg.begin(), gvecg.end(), pout+5*m*n+1);
}



////////////////////////////////////////////////////////////////////////////////////////////////////
// half_peak_width
// 
template <typename InputVector, typename OutputVector>
void half_peak_width(size_t m, size_t n, const InputVector& ldiff, const InputVector& idata, OutputVector& data)
{
  typedef typename InputVector::value_type T;

  thrust::device_vector<Numeric> peakmask(m*n);
  thrust::transform(idata.begin(), idata.end(), peakmask.begin(), mask01<Numeric>(Integer(0)));
  //max for peakmask
  thrust::equal_to<Numeric> binary_pred;
  thrust::maximum<Numeric>  binary_max;
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), idata.begin(), data.begin(),binary_pred,binary_max);
  thrust::reverse(data.begin(), data.end());
  thrust::reverse(peakmask.begin(), peakmask.end());
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), data.begin(), data.begin(),binary_pred,binary_max);
  thrust::reverse(data.begin(), data.end());
  thrust::device_vector<Numeric> halfdata(m*n);
  thrust::fill(halfdata.begin(),halfdata.end(),2);
  thrust::transform(data.begin(), data.end(), halfdata.begin(), data.begin(), thrust::divides<Numeric>());
  //minus half max
  thrust::transform(idata.begin(), idata.end(), data.begin(), halfdata.begin(), thrust::minus<Numeric>());
  thrust::transform(halfdata.begin(), halfdata.end(), peakmask.begin(), mask01<Numeric>(Integer(0)));
  thrust::plus<Numeric>  binary_plus;
  //width in mask
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), ldiff.begin(), data.begin(),binary_pred,binary_plus);
  thrust::reverse(data.begin(), data.end());
  thrust::reverse(peakmask.begin(), peakmask.end());
  thrust::inclusive_scan_by_key(peakmask.begin(), peakmask.end(), data.begin(), data.begin(),binary_pred,binary_max);
  thrust::reverse(data.begin(), data.end());
}



////////////////////////////////////////////////////////////////////////////////////////////////////
// find_doublemaxpeaks_lambdahalf - a gpu function for 
//
template <typename InputVector, typename OutputVector>
void find_doublemaxpeaks_lambdahalf(size_t m, size_t n, const InputVector& lambda, const InputVector& idata, const Numeric& w1, const Numeric& w2, const Numeric& w3, 
OutputVector& pout, OutputVector& avgbasoffp, OutputVector& pintegralp, OutputVector& halfwp, OutputVector& dout, OutputVector& avgbasoffd, OutputVector& pintegrald, OutputVector& halfwd, const Numeric& navg)
{
	thrust::device_vector<Numeric> ldiff(m*n);
	thrust::transform(lambda.begin()+1, lambda.end(), lambda.begin(), ldiff.begin(), thrust::minus<Numeric>());

	thrust::device_vector<Numeric> orgbasoffp(m*n);
	thrust::device_vector<Numeric> orgbasoffd(m*n);

    thrust::device_vector<Numeric> data(m*n);
    thrust::device_vector<Numeric> data2(m*n);

    double_moving_average(m,n,idata, w2, data2);
    thrust::transform(idata.begin(), idata.end(), data2.begin(), data.begin(), thrust::divides<Numeric>());

    //original peaks up and down
    thrust::transform(data.begin(), data.end(), orgbasoffp.begin(), oneup<Numeric>(Numeric(1)));
    thrust::transform(data.begin(), data.end(), orgbasoffd.begin(), onedown<Numeric>(Numeric(1)));

    thrust::equal_to<Numeric> binary_pred;
    thrust::maximum<Numeric>  binary_max;
 
    double_moving_average(m,n,idata, w1, data);
    thrust::transform(data.begin(), data.end(), data2.begin(), data.begin(), thrust::divides<Numeric>());
	//average peaks up and down
    thrust::transform(data.begin(), data.end(), avgbasoffp.begin(), oneup<Numeric>(Numeric(1)));
    thrust::transform(data.begin(), data.end(), avgbasoffd.begin(), onedown<Numeric>(Numeric(1)));

	//the same number within each vector
    thrust::device_vector<Integer> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end(),0);
    thrust::transform(vindex.begin(), vindex.end(), vindex.begin(), kindcreate(Integer(m),Integer(n)));

    //orgbasoffp, avgbasoffp
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoffp.begin(), data.begin(), binary_pred,thrust::plus<Numeric>());
    thrust::reverse(data.begin(), data.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), data.begin(), data2.begin(),binary_pred,binary_max);
    thrust::reverse(data2.begin(), data2.end());
    thrust::fill(data.begin(),data.end(),m/(navg/w1));
    thrust::transform(data2.begin(), data2.end(), data.begin(), data2.begin(), thrust::divides<Numeric>());

    Numeric2Iterator first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffp.begin(), data2.begin()));
    Numeric2Iterator last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffp.end(),   data2.end()));

    thrust::transform(first, last, pout.begin(), zipup());

	//double_moving_average(m,n,data, w3, pout);

    thrust::transform(pout.begin()+1, pout.end(), pout.begin(), data.begin(), thrust::minus<Numeric>());
thrust::copy(data.begin(), data.end(), avgbasoffp.begin());    
    Numeric2Iterator first0 = thrust::make_zip_iterator(thrust::make_tuple(data.begin(), data2.begin()));
    Numeric2Iterator first1 = thrust::make_zip_iterator(thrust::make_tuple(data.begin() + 1, data2.begin() + 1));
    Numeric2Iterator last0  = thrust::make_zip_iterator(thrust::make_tuple(data.end(),  data2.end()));

    thrust::transform(first0, last0, first1, pout.begin(), compare_zip());

    //orgbasoffd, avgbasoffd
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoffd.begin(), data.begin(), binary_pred,thrust::plus<Numeric>());
    thrust::reverse(data.begin(), data.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), data.begin(), data2.begin(),binary_pred,binary_max);
    thrust::reverse(data2.begin(), data2.end());
    thrust::fill(data.begin(),data.end(),m/(navg/w1));
    thrust::transform(data2.begin(), data2.end(), data.begin(), data2.begin(), thrust::divides<Numeric>());

    first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffd.begin(), data2.begin()));
    last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffd.end(),   data2.end()));

    thrust::transform(first, last, dout.begin(), zipup());

    //double_moving_average(m,n,data, w3, dout);

    thrust::transform(dout.begin()+1, dout.end(), dout.begin(), data.begin(), thrust::minus<Numeric>());
thrust::copy(data.begin(), data.end(), avgbasoffd.begin());
    first0 = thrust::make_zip_iterator(thrust::make_tuple(data.begin(), data2.begin()));
    first1 = thrust::make_zip_iterator(thrust::make_tuple(data.begin() + 1, data2.begin() + 1));
    last0  = thrust::make_zip_iterator(thrust::make_tuple(data.end(),  data2.end()));
    
    thrust::transform(first0, last0, first1, dout.begin(), compare_zip());
    //lambda_peak_sums(m, n, ldiff, avgbasoffp, pintegralp);
    //lambda_peak_sums(m, n, ldiff, avgbasoffd, pintegrald);
    //half_peak_width(m, n, ldiff, avgbasoffp, halfwp);
    //half_peak_width(m, n, ldiff, avgbasoffd, halfwd);
}



////////////////////////////////////////////////////////////////////////////////////////////////////
// gprpeaklambda2halfmask - the host-gpu function for 
//
void gprpeaklambda2halfmask(PNumeric lint, PNumeric pint, PInteger a, PInteger b, PNumeric win1, PNumeric win2, PNumeric win3, PNumeric pout, PInteger navg) {
    size_t m = a[0];//row number
    size_t n = b[0];//column number
    int w1 = win1[0];//window
    int w2 = win2[0];//baseline window
    int w3 = win3[0];//difference window
	int navg0 = navg[0];

    // transfer data to the device
    thrust::device_vector<Numeric> lambda(lint,lint+a[0]*b[0]);
    thrust::device_vector<Numeric> gveca(pint,pint+a[0]*b[0]);
    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecc(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecd(a[0]*b[0]);
    thrust::device_vector<Numeric> gvece(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecf(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecg(a[0]*b[0]);
    thrust::device_vector<Numeric> gvech(a[0]*b[0]);
    thrust::device_vector<Numeric> gveci(a[0]*b[0]);
    
    find_doublemaxpeaks_lambdahalf(m,n,lambda,gveca,w1,w2,w3,gvecb,gvecc,gvecd,gvece,gvecf,gvecg,gvech,gveci,navg0);
    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
    thrust::copy(gvecc.begin(), gvecc.end(), pout+m*n+1);
    thrust::copy(gvecd.begin(), gvecd.end(), pout+2*m*n+1);
    thrust::copy(gvece.begin(), gvece.end(), pout+3*m*n+1);
    thrust::copy(gvecf.begin(), gvecf.end(), pout+4*m*n+1);
    thrust::copy(gvecg.begin(), gvecg.end(), pout+5*m*n+1);
    thrust::copy(gvech.begin(), gvech.end(), pout+6*m*n+1);
    thrust::copy(gveci.begin(), gveci.end(), pout+7*m*n+1);
}


////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////
// exemplary gpu functors
//
template <typename T>
struct is_less_than_zero
{
   __host__ __device__
   bool operator()(T x)
   {
      return x < 0;
   }
};

template <typename T>
struct is_greater_than
{
   T w;
   __host__ __device__
   is_greater_than(T w) : w(w) {}
   __host__ __device__
   bool operator()(T x)
   {
      return x > w;
   }
};


template <typename T>
struct minus_and_divide : public thrust::binary_function<T,T,T>
{
    T w;

    minus_and_divide(T w) : w(w) {}

    __host__ __device__
    T operator()(const T& a, const T& b) const
    {
        return (a - b) / w;
    }
};

template <typename T>
struct minus_and_divide_w : public thrust::binary_function<T,T,T>
{
    T w, maxb;

    __host__ __device__
    minus_and_divide_w(T w, T maxb) : w(w), maxb(maxb) {}

    __host__ __device__
    T operator()(const T& a, const T& b) const
    {
    	if(a < b)
        return (a - b) / w;
	else 
	return (a - maxb) / (w - a % maxb);
    }
};

// convert a linear index to a linear index in the transpose 
struct transpose_index : public thrust::unary_function<size_t,size_t>
{
    size_t m, n;

    __host__ __device__
    transpose_index(size_t _m, size_t _n) : m(_m), n(_n) {}

    __host__ __device__
    size_t operator()(size_t linear_index)
    {
        size_t j = linear_index / m;
        size_t i = linear_index % m;

        return j + i * n;
    }
};

// convert a linear index to a row index
struct column_index : public thrust::unary_function<size_t,size_t>
{
    size_t n;
    
    __host__ __device__
    column_index(size_t _n) : n(_n) {}

    __host__ __device__
    size_t operator()(size_t i)
    {
        return i / n;
    }
};

// convert a linear index to a row index
struct binary_index : public thrust::unary_function<size_t,size_t>
{
    size_t n;
    
    __host__ __device__
    binary_index(size_t _n) : n(_n) {}

    __host__ __device__
    size_t operator()(size_t i)
    {
        return (i / n) % 2;
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////
// exemplary gpu functions
//

// transpose an M-by-N array
template <typename T>
void transpose(size_t m, size_t n, thrust::device_vector<T>& src, thrust::device_vector<T>& dst)
{
    thrust::counting_iterator<size_t> indices(0);
    
    thrust::gather(thrust::make_transform_iterator(indices, transpose_index(n, m)),
                   thrust::make_transform_iterator(indices, transpose_index(n, m)) + dst.size(),
                   src.begin(),
                   dst.begin());
}


// scan the rows of an M-by-N array
template <typename T>
void scan_horizontally(size_t m, size_t n, thrust::device_vector<T>& d_data)
{
    thrust::counting_iterator<size_t> indices(0);

    thrust::inclusive_scan_by_key(d_data.begin(), d_data.end(),
                                  thrust::make_transform_iterator(indices, column_index(m)),
                                  d_data.begin());
}




template <typename T>
void sumvec(thrust::device_vector<T>& gvec, Numeric& out)
{
    thrust::reduce(gvec.begin(), gvec.end(), out);
}

